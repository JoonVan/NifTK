#include "hip/hip_runtime.h"
/*=============================================================================

  NifTK: A software platform for medical image computing.

  Copyright (c) University College London (UCL). All rights reserved.

  This software is distributed WITHOUT ANY WARRANTY; without even
  the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR
  PURPOSE.

  See LICENSE.txt in the top level directory for details.

=============================================================================*/

#include "niftkAddTwoIntegers.h"
#include <niftkCUDAUtils.h>

namespace niftk
{

//-----------------------------------------------------------------------------
__global__ void add_two_integers(int *a, int *b, int *c)
{
  *c = *a + *b;
}


//-----------------------------------------------------------------------------
int AddTwoIntegers(int a, int b)
{
  int c;
  int *dev_a, *dev_b, *dev_c;
  int size = sizeof( int );

  niftkCUDACall( hipMalloc( (void**)&dev_a, size ));
  niftkCUDACall( hipMalloc( (void**)&dev_b, size ));
  niftkCUDACall( hipMalloc( (void**)&dev_c, size ));

  niftkCUDACall( hipMemcpy( dev_a, &a, size, hipMemcpyHostToDevice ));
  niftkCUDACall( hipMemcpy( dev_b, &b, size, hipMemcpyHostToDevice ));

  add_two_integers<<< 1, 1 >>>( dev_a, dev_b, dev_c );

  niftkCUDACall( hipMemcpy( &c, dev_c, size, hipMemcpyDeviceToHost ));

  niftkCUDACall( hipFree( dev_a ));
  niftkCUDACall( hipFree( dev_b ));
  niftkCUDACall( hipFree( dev_c ));

  return c;
}

} // end namespace

#include "hip/hip_runtime.h"
/*=============================================================================

  libvideo: a library for SDI video processing.

  Copyright (c) University College London (UCL). All rights reserved.

  This software is distributed WITHOUT ANY WARRANTY; without even
  the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR
  PURPOSE.

  See LICENSE.txt in the top level directory for details.

=============================================================================*/

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_texture_types.h>


texture<uchar4, hipTextureType2D, hipReadModeNormalizedFloat>     rgbatex;


__global__
void rgba2nv12_kernel(char* dst, std::size_t dstpitch, int width, int height, int paddedheight)
{
    int xid = blockIdx.x * blockDim.x + threadIdx.x;
    int yid = blockIdx.y * blockDim.y + threadIdx.y;

    if ((xid < width) && (yid < height))
    {
        float4  rgba = tex2D(rgbatex, xid, height - yid - 1);

        // FIXME: no idea *where* the chroma samles are supposed to be
        //        do i need to average the rgba samples and then convert?
        //        or do i just subsample?
        // FIXME: pick average from mipmap


        float   y = 0.299f * rgba.x + 0.587f * rgba.y + 0.114f * rgba.z;
        float   u = (rgba.z - y) * 0.565f;
        float   v = (rgba.x - y) * 0.713f;

        u += 0.5f;
        v += 0.5f;

        // notice the multiplication with 255: incomding is normalised texture read but outgoing is global mem write
        dst[yid * dstpitch + xid] = y * 255;

        if (((xid % 2) == 0) && ((yid % 2) == 0))
        {
            dst[paddedheight * dstpitch + yid/2 * dstpitch + xid] = u * 255;
            dst[paddedheight * dstpitch + yid/2 * dstpitch + xid+1] = v * 255;
        }
    }
}

extern "C"
bool rgba2nv12(char* dst, std::size_t dstpitch, hipArray_t array, int width, int height, int paddedheight)
{
    hipChannelFormatDesc   c = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);
    hipError_t error = hipBindTextureToArray(&rgbatex, array, &c);
    if (error != hipSuccess)
        return false;

    // all relevant widths are dividable by 16
    // height however is not
    const int   TILE_DIM = 16;
    dim3    grid((width + 15) / TILE_DIM, (height + 15) / TILE_DIM);
    dim3    threads(TILE_DIM, TILE_DIM);

    rgba2nv12_kernel<<<grid, threads>>>(dst, dstpitch, width, height, paddedheight);

    return true;
}
